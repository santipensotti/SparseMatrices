#include "helpers/mtxToCuda.h"
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <cstdlib>
#include <chrono>
#include <fstream>
#include <string>
#include <Eigen/Sparse>

#define CHECK_CUDA(x)  do { hipError_t err = (x); if (err != hipSuccess) { \
  fprintf(stderr, "CUDA %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); exit(1);} } while(0)
#define CHECK_CUSPARSE(x) do { hipsparseStatus_t st = (x); if (st != HIPSPARSE_STATUS_SUCCESS) { \
  fprintf(stderr, "cuSPARSE %s:%d: %d\n", __FILE__, __LINE__, (int)st); exit(1);} } while(0)

template<typename T>
struct descriptorDevice {
    int rows=0, cols=0, nnz=0;
    int *d_rowptr=nullptr, *d_colind=nullptr;
    T   *d_vals=nullptr, *d_x=nullptr;
};

template<typename T>
descriptorDevice<T> to_device(const CSRHost<T>& H, const std::vector<T>& x) {
    descriptorDevice<T> D{H.rows, H.cols, H.nnz};
    CHECK_CUDA(hipMalloc((void**)&D.d_rowptr, (size_t)(H.rows+1)*sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&D.d_colind, (size_t)H.nnz*sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&D.d_vals,   (size_t)H.nnz*sizeof(T)));
    CHECK_CUDA(hipMemcpy(D.d_rowptr, H.rowptr.data(), (size_t)(H.rows+1)*sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(D.d_colind, H.colind.data(), (size_t)H.nnz*sizeof(int),      hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(D.d_vals,   H.vals.data(),   (size_t)H.nnz*sizeof(T),        hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc((void**)&D.d_x, (size_t)x.size()*sizeof(T)));
    CHECK_CUDA(hipMemcpy(D.d_x, x.data(), (size_t)x.size()*sizeof(T), hipMemcpyHostToDevice));
    return D;
}

template<typename T>
void destroy_device(descriptorDevice<T>& D) {
    if (D.d_rowptr) hipFree(D.d_rowptr);
    if (D.d_colind) hipFree(D.d_colind);
    if (D.d_vals)   hipFree(D.d_vals);
    if (D.d_x)      hipFree(D.d_x);
    D = {};
}

double runSpMV_once(hipsparseHandle_t handle,
                    const hipsparseSpMatDescr_t descr_A,
                    int m, int n,
                    const float* dx, float* dy,
                    bool is_coo=false) {
    hipsparseDnVecDescr_t x_vec=nullptr, y_vec=nullptr;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&x_vec, (int64_t)n, (void*)dx, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&y_vec, (int64_t)m, (void*)dy, HIP_R_32F));
    float alpha = 1.0f, beta = 0.0f;
    const hipsparseSpMVAlg_t alg = is_coo ? HIPSPARSE_SPMV_COO_ALG1
                                         : HIPSPARSE_SPMV_CSR_ALG1;

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descr_A, x_vec, &beta, y_vec, HIP_R_32F, alg, &bufferSize));

    void* dBuffer = nullptr;
    if (bufferSize > 0) CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));


    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    const int iters = 1;
    const int warm_up = 0;
    for (int i=0; i < warm_up; ++i){
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, descr_A, x_vec, &beta, y_vec, HIP_R_32F, alg, dBuffer));
    }
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < iters; ++i) {
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, descr_A, x_vec, &beta, y_vec, HIP_R_32F, alg, dBuffer));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float total_ms=0.0f; CHECK_CUDA(hipEventElapsedTime(&total_ms, start, stop));
    double ms = total_ms / iters;

    if (dBuffer) hipFree(dBuffer);
    hipEventDestroy(start); hipEventDestroy(stop);
    hipsparseDestroyDnVec(x_vec); hipsparseDestroyDnVec(y_vec);
    return ms;
}

void append_row_csv(const std::string& csv_path, const std::string& matrix_name, int n, int k, int nnz, double t_csr, double t_coo, double t_eigen) {
    bool write_header = true;
    {
        std::ifstream test(csv_path, std::ios::binary);
        if (test.good()) { test.seekg(0, std::ios::end); write_header = (test.tellg() == 0); }
    }
    std::ofstream f(csv_path, std::ios::app);
    if (!f) { fprintf(stderr, "No pude abrir %s\n", csv_path.c_str()); return; }
    if (write_header) f << "matrix_name,n,k,nnz,tiempoCSR_ms,tiempoCOO_ms,tiempoEigen_ms\n";
    f << matrix_name << "," << n << "," << k << "," << nnz << ","
      << std::fixed << t_csr << "," << t_coo << "," << t_eigen << "\n";
}

int main(int argc, char** argv) {
    const char* path = (argc > 1 ? argv[1] : "example.mtx");
    Eigen::initParallel();
    using T = float;
    CSRHost<T> H = load_csr_from_mtx<T>(path);
    printf("Matriz: %d x %d  nnz=%d\n", H.rows, H.cols, H.nnz);

    std::vector<T> hx(H.cols), hy(H.rows, 0.0f);
    std::srand(42);
    for (int i = 0; i < (int)hx.size(); ++i) hx[i] = (T)std::rand() / RAND_MAX;
    descriptorDevice<T> dA = to_device(H, hx);

    T *dy=nullptr, *dy_2=nullptr;
    CHECK_CUDA(hipMalloc((void**)&dy,   (size_t)H.rows*sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&dy_2, (size_t)H.rows*sizeof(T)));
    CHECK_CUDA(hipMemset(dy,   0, (size_t)H.rows*sizeof(T)));
    CHECK_CUDA(hipMemset(dy_2, 0, (size_t)H.rows*sizeof(T)));

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // CSR descriptor
    hipsparseSpMatDescr_t A_csr;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &A_csr, (int64_t)H.rows, (int64_t)H.cols, (int64_t)H.nnz, 
        (void*)dA.d_rowptr, (void*)dA.d_colind, (void*)dA.d_vals,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    double time_csr = runSpMV_once(handle, A_csr, H.rows, H.cols, dA.d_x, dy, false);
    CHECK_CUSPARSE(hipsparseDestroySpMat(A_csr));

    // CSR -> COO rows
    int *A_coo_rows = nullptr;
    CHECK_CUDA(hipMalloc(&A_coo_rows, (size_t)H.nnz * sizeof(int)));
    CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA.d_rowptr, H.nnz, H.rows, A_coo_rows, HIPSPARSE_INDEX_BASE_ZERO));

    // COO descriptor
    hipsparseSpMatDescr_t A_coo;
    CHECK_CUSPARSE(hipsparseCreateCoo(&A_coo, (int64_t)H.rows, (int64_t)H.cols, (int64_t)H.nnz,
        A_coo_rows, (void*)dA.d_colind, (void*)dA.d_vals,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    double time_coo = runSpMV_once(handle, A_coo, H.rows, H.cols, dA.d_x, dy_2, true);
    CHECK_CUSPARSE(hipsparseDestroySpMat(A_coo));
    CHECK_CUDA(hipFree(A_coo_rows));

    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(hy.data(), dy, (size_t)H.rows*sizeof(T), hipMemcpyDeviceToHost));

    // CPU Eigen
    Eigen::setNbThreads(1);
    Eigen::SparseMatrix<T, Eigen::ColMajor, int> A = load_eigen_from_mtx<T>(path);
    A.makeCompressed();
    Eigen::Map<const Eigen::Matrix<T, Eigen::Dynamic, 1>> x_cpu(hx.data(), A.cols());
    auto t0 = std::chrono::high_resolution_clock::now();
    Eigen::Matrix<T, Eigen::Dynamic, 1> y_cpu = A * x_cpu;
    auto t1 = std::chrono::high_resolution_clock::now();
    double ms_cpu = std::chrono::duration<double, std::milli>(t1 - t0).count();

    // Limpieza
    CHECK_CUDA(hipFree(dy));
    CHECK_CUDA(hipFree(dy_2));
    destroy_device(dA);
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    append_row_csv("resultados.csv", path, H.rows, H.cols, H.nnz, time_csr, time_coo, ms_cpu);
    return 0;
}
