#include "hip/hip_runtime.h"
#include "helpers/mtxToCuda.h"          // Debe proveer CSRHost<T> y load_csr_from_mtx<T>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <cstdlib>
#include <chrono>
#include <fstream>
#include <string>
#include <Eigen/Sparse>

/*
The function performs the multiplication of a sparse matrix matA and a dense matrix matB.
where
op(A) is a sparse matrix of size 
op(B) is a dense matrix of size 
C is a dense matrix of size 
*/

// ---- Chequeo de errores (CUDA + cuSPARSE) ----
#define CHECK_CUDA(x)  do { hipError_t err = (x); if (err != hipSuccess) { \
  fprintf(stderr, "CUDA %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); exit(1);} } while(0)

#define CHECK_CUSPARSE(x) do { hipsparseStatus_t st = (x); if (st != HIPSPARSE_STATUS_SUCCESS) { \
  fprintf(stderr, "cuSPARSE %s:%d: %d\n", __FILE__, __LINE__, (int)st); exit(1);} } while(0)

// ---- Contenedor device para CSR ----
template<typename T>
struct descriptorDevice {
    int rows=0, cols=0, nnz=0;
    int *d_rowptr=nullptr, 
    *d_colind=nullptr;
    T* d_vals=nullptr;
    T* d_B=nullptr; // Matriz densa B
    int B_cols = 0; // Numero de columnas de B
};



template<typename T>
descriptorDevice<T> to_device(const CSRHost<T>& H, const std::vector<T>& B, int B_cols) {
    descriptorDevice<T> D{H.rows, H.cols, H.nnz};
    D.B_cols = B_cols;
    CHECK_CUDA(hipMalloc((void**)&D.d_rowptr, (H.rows+1)*sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&D.d_colind, H.nnz*sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&D.d_vals,   H.nnz*sizeof(T)));
    CHECK_CUDA(hipMemcpy(D.d_rowptr, H.rowptr.data(), (H.rows+1)*sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(D.d_colind, H.colind.data(), H.nnz*sizeof(int),      hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(D.d_vals,   H.vals.data(),   H.nnz*sizeof(T),        hipMemcpyHostToDevice));

    // Allocate and copy B matrix (type T)
    CHECK_CUDA(hipMalloc((void**)&D.d_B, B.size()*sizeof(T)));
    CHECK_CUDA(hipMemcpy(D.d_B, B.data(), B.size()*sizeof(T), hipMemcpyHostToDevice));

    return D;
}

template<typename T>
void destroy_device(descriptorDevice<T>& D) {
    if (D.d_rowptr) hipFree(D.d_rowptr);
    if (D.d_colind) hipFree(D.d_colind);
    if (D.d_vals)   hipFree(D.d_vals);
    if (D.d_B)     hipFree(D.d_B);
    D = {};
}

double runSpMM(const descriptorDevice<float>& dA, float* dC, hipsparseSpMatDescr_t& descr_A, const bool is_coo=false) {
    const int m = dA.rows;
    const int n = dA.cols;
    const int k = dA.B_cols;

    hipsparseHandle_t handle;  CHECK_CUSPARSE(hipsparseCreate(&handle));
    
    hipsparseDnMatDescr_t B_mat, C_mat;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&B_mat, n, k, n, (void*)dA.d_B, HIP_R_32F, HIPSPARSE_ORDER_COL));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&C_mat, m, k, m, (void*)dC, HIP_R_32F, HIPSPARSE_ORDER_COL));

    float alpha = 1.0f, beta = 0.0f;

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descr_A, B_mat, &beta, C_mat,
        HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT,
        &bufferSize
    ));

    void* dBuffer = nullptr;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));
    const int iters_warmup = 2, iters = 10;
    for (int i = 0; i < iters_warmup; ++i) {
        CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, descr_A, B_mat, &beta, C_mat, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));
    }
    CHECK_CUDA(hipDeviceSynchronize());
    
    float total_ms = 0.0f;
    for (int i = 0; i < iters; ++i) {
        hipEvent_t s,e; CHECK_CUDA(hipEventCreate(&s)); CHECK_CUDA(hipEventCreate(&e));
        CHECK_CUDA(hipEventRecord(s));
        CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, descr_A, B_mat, &beta, C_mat, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));
        CHECK_CUDA(hipEventRecord(e));
        CHECK_CUDA(hipEventSynchronize(e));
        float ms=0; CHECK_CUDA(hipEventElapsedTime(&ms, s, e));
        total_ms += ms;
        CHECK_CUDA(hipEventDestroy(s)); CHECK_CUDA(hipEventDestroy(e));
    }
    hipFree(dBuffer);
    hipsparseDestroyDnMat(B_mat);
    hipsparseDestroyDnMat(C_mat);
    hipsparseDestroy(handle);
    double ms = total_ms / iters;
    return ms;
}


void append_row_csv(const std::string& csv_path, const std::string& matrix_name, int m, int n, int k, int nnz, double t_csr, double t_coo, double t_eigen) {
    bool write_header = true;
    {
        std::ifstream test(csv_path, std::ios::binary);
        if (test.good()) {
            test.seekg(0, std::ios::end);
            write_header = (test.tellg() == 0);
        } else {
            write_header = true; // no existe -> escribir cabecera
        }
    }
    std::ofstream f(csv_path, std::ios::app);
    if (!f) { fprintf(stderr, "No pude abrir %s\n", csv_path.c_str()); return; }
    if (write_header) {
        f << "matrix_name,m,n,k,nnz,tiempoCSR_ms,tiempoCOO_ms,tiempoEigen_ms\n";
    }
    f << matrix_name << "," << m << "," << n << "," << k << "," << nnz << ","
      << std::fixed << t_csr << "," << t_coo << "," << t_eigen << "\n";
}

template<typename T>
void errorGpuCpu(const Eigen::Matrix<T, -1, -1>& C_cpu,
                 const Eigen::Matrix<T, -1, -1>& C_gpu) {
    double abs_l2 = (C_cpu.template cast<double>() - C_gpu.template cast<double>()).norm();
    double rel_l2 = abs_l2 / (C_cpu.template cast<double>().norm() + 1e-30);
    double linf    = (C_cpu - C_gpu).template lpNorm<Eigen::Infinity>();

    printf("abs L2=%.3e  rel L2=%.3e  L_inf=%.3e\n", abs_l2, rel_l2, linf);
}

int main(int argc, char** argv) {
    const char* path = (argc > 1 ? argv[1] : "example.mtx");


    // 1) Cargar CSR (host)
    using T = float;
    CSRHost<T> H = load_csr_from_mtx<T>(path);
    printf("Matriz: %d x %d  nnz=%d\n", H.rows, H.cols, H.nnz);
    int k = H.rows;
    if (H.nnz == 0) return 1;
    // 3) Crear B y C en host
    std::vector<T> hB(H.cols * k);
    std::vector<T> hC(H.rows * k, 0.0f);
    std::srand(42);
    for (int i = 0; i < (int)hB.size(); ++i) hB[i] = (T)std::rand() / RAND_MAX;
    
    descriptorDevice<T> dA = to_device(H, hB, k);

    // 4) Reservar C en device y copiar
    T *dC=nullptr, *dC_2=nullptr; 
    CHECK_CUDA(hipMalloc((void**)&dC, hC.size()*sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&dC_2, hC.size()*sizeof(T)));
    CHECK_CUDA(hipMemcpy(dC, hC.data(), hC.size()*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dC_2, hC.data(), hC.size()*sizeof(T), hipMemcpyHostToDevice));

    // 5) Ejecutar SpMM con Csr
    printf("Ejecutando SpMM...\n");
    const int m = dA.rows, n = dA.cols, nnz = dA.nnz;    

    hipsparseSpMatDescr_t A_csr;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &A_csr, m, n, nnz,
        (void*)dA.d_rowptr, (void*)dA.d_colind, (void*)dA.d_vals,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F
    ));
    
    double time_csr = runSpMM(dA, dC, A_csr);
    printf("SpMM completado.\n");
    CHECK_CUSPARSE(hipsparseDestroySpMat(A_csr));

    // 5) COO
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    int *A_coo_rows = nullptr;
    CHECK_CUDA(hipMalloc(&A_coo_rows, nnz * sizeof(int)));
    CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA.d_rowptr, nnz, m,
                           A_coo_rows, HIPSPARSE_INDEX_BASE_ZERO));
    
    hipsparseSpMatDescr_t A_coo;
    CHECK_CUSPARSE(hipsparseCreateCoo(&A_coo, m, n, nnz,
                            A_coo_rows, (void*)dA.d_colind, (void*)dA.d_vals,
                            HIPSPARSE_INDEX_32I,
                            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    
    double time_coo = runSpMM(dA, dC_2, A_coo, true);
    CHECK_CUSPARSE(hipsparseDestroySpMat(A_coo));

    // 6) Traer resultado y mostrar algunas entradas
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(hC.data(), dC, hC.size()*sizeof(T), hipMemcpyDeviceToHost));
    
    // 7) Cargar Matrix Eigen 
    Eigen::SparseMatrix<T, Eigen::ColMajor, int> A = load_eigen_from_mtx<T>(path);
    A.makeCompressed();
    Eigen::Map<const Eigen::Matrix<T, Eigen::Dynamic, Eigen::Dynamic, Eigen::ColMajor>> B_cpu(hB.data(), A.cols(), k);

    // 8) Medir tiempo de SpMV en CPU (Eigen)
    auto t0 = std::chrono::high_resolution_clock::now();
    Eigen::Matrix<T, Eigen::Dynamic, Eigen::Dynamic, Eigen::ColMajor> C_cpu = A * B_cpu;
    auto t1 = std::chrono::high_resolution_clock::now();
    double ms_cpu = std::chrono::duration<double, std::milli>(t1 - t0).count();
    
    // 9) Comparar error Cpu y Gpu (Csr y Coo)
    Eigen::Map<const Eigen::Matrix<T, -1, -1, Eigen::ColMajor>> C_gpu(hC.data(), m, k);
    CHECK_CUDA(hipMemcpy(hC.data(), dC_2, hC.size()*sizeof(T), hipMemcpyDeviceToHost));
    Eigen::Map<const Eigen::Matrix<T, -1, -1, Eigen::ColMajor>> C_gpu_2(hC.data(), m, k);

    // errorGpuCpu(C_cpu, C_gpu);  
    // errorGpuCpu(C_cpu, C_gpu_2);

    // 10) Limpieza

    CHECK_CUDA(hipFree(dC));
    CHECK_CUDA(hipFree(dC_2));
    destroy_device(dA);
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    append_row_csv("SpMM_results.csv", path, m, n, k, nnz, time_csr, time_coo, ms_cpu);
    return 0;
}

