#include <cstdio>
#include <stdexcept>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsparse.h>


#define CHECK(status) do { _check((status), __FILE__, __LINE__); } while(false)
inline void _check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d %s: %s. In file '%s' on line %d\n", error_code, hipGetErrorName(error_code), hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(1);
    }
}
inline void _check(hipsparseStatus_t status, const char *file, int line)
{
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        fprintf(stderr, "CUSPARSE Error %d %s: %s. In file '%s' on line %d\n", status, hipsparseGetErrorName(status), hipsparseGetErrorString(status), file, line);
        fflush(stderr);
        exit(1);
    }
}



struct trsm_system
{
    int size;
    int nrhs;
    int nnz;
    int ld;
    int * A_rowptrs = nullptr;
    int * A_colidxs = nullptr;
    double * A_vals = nullptr;
    double * B = nullptr; // row-major
    double * X = nullptr; // row-major
};

trsm_system load_system(const char * file)
{
    FILE * f = fopen(file, "r");
    if(f == nullptr) throw std::runtime_error("cannot open matrix file");

    trsm_system sys;
    fscanf(f, "%d%d%d", &sys.size, &sys.nrhs, &sys.nnz);
    std::vector<int> rowptrs(sys.size+1);
    std::vector<int> colidxs(sys.nnz);
    std::vector<double> vals(sys.nnz);
    for(int r = 0; r <= sys.size; r++) fscanf(f, "%d", &rowptrs[r]);
    for(int i = 0; i < sys.nnz; i++) fscanf(f, "%d", &colidxs[i]);
    for(int i = 0; i < sys.nnz; i++) fscanf(f, "%lf", &vals[i]);

    fclose(f);

    std::vector<double> B(sys.size * sys.nrhs);
    srand(42); // Set seed for reproducibility
    for(int i = 0; i < B.size(); i++) B[i] = (double)rand() / RAND_MAX;

    CHECK(hipMalloc(&sys.A_rowptrs, (sys.size + 1) * sizeof(int)));
    CHECK(hipMalloc(&sys.A_colidxs, sys.nnz * sizeof(int)));
    CHECK(hipMalloc(&sys.A_vals, sys.nnz * sizeof(double)));
    size_t pitch_B;
    CHECK(hipMallocPitch(&sys.B, &pitch_B, sys.nrhs * sizeof(double), sys.size));
    sys.ld = pitch_B / sizeof(double);
    size_t pitch_X;
    CHECK(hipMallocPitch(&sys.X, &pitch_X, sys.nrhs * sizeof(double), sys.size));
    if(pitch_X != pitch_B) throw std::runtime_error("different pitches");

    CHECK(hipMemcpy(sys.A_rowptrs, rowptrs.data(), (sys.size + 1) * sizeof(int), hipMemcpyDefault));
    CHECK(hipMemcpy(sys.A_colidxs, colidxs.data(), sys.nnz * sizeof(int), hipMemcpyDefault));
    CHECK(hipMemcpy(sys.A_vals, vals.data(), sys.nnz * sizeof(double), hipMemcpyDefault));
    CHECK(hipMemcpy2D(sys.B, pitch_B, B.data(), sys.nrhs * sizeof(double), sys.nrhs * sizeof(double), sys.size, hipMemcpyDefault));

    return sys;
}


void run_spm(const trsm_system& sys, hipsparseSpMatDescr_t& descr_A, const hipsparseHandle_t& handle, bool is_coo = false){
    hipsparseDnMatDescr_t descr_B, descr_X;
    CHECK(hipsparseCreateDnMat(&descr_B, sys.size, sys.nrhs, sys.ld, sys.B, HIP_R_64F, HIPSPARSE_ORDER_ROW));
    CHECK(hipsparseCreateDnMat(&descr_X, sys.size, sys.nrhs, sys.ld, sys.X, HIP_R_64F, HIPSPARSE_ORDER_ROW));

    hipsparseSpSMDescr_t descr_spsm;
    CHECK(hipsparseSpSM_createDescr(&descr_spsm));

    hipEvent_t e_start, e_stop;
    CHECK(hipEventCreate(&e_start));
    CHECK(hipEventCreate(&e_stop));

    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    double one = 1.0;
    size_t buffersize;
    void * buffer;
    int warmup = 2;
    int repeats = 10;
    float ms_total = 0;

    CHECK(hipsparseSpSM_bufferSize(handle, opA, opB, &one, descr_A, descr_B, descr_X, HIP_R_64F, HIPSPARSE_SPSM_ALG_DEFAULT, descr_spsm, &buffersize));
    CHECK(hipDeviceSynchronize());
    CHECK(hipMalloc(&buffer, buffersize));

    CHECK(hipsparseSpSM_analysis(handle, opA, opB, &one, descr_A, descr_B, descr_X, HIP_R_64F, HIPSPARSE_SPSM_ALG_DEFAULT, descr_spsm, buffer));
    CHECK(hipDeviceSynchronize());

    for(int rep = 0; rep < warmup + repeats; rep++)
    {
        CHECK(hipEventRecord(e_start));
        CHECK(hipsparseSpSM_solve(handle, opA, opB, &one, descr_A, descr_B, descr_X, HIP_R_64F, HIPSPARSE_SPSM_ALG_DEFAULT, descr_spsm, nullptr));
        CHECK(hipEventRecord(e_stop));
        CHECK(hipDeviceSynchronize());
        float ms;
        CHECK(hipEventElapsedTime(&ms, e_start, e_stop));

        if(rep > warmup) ms_total += ms;
    }

    CHECK(hipFree(buffer));
    CHECK(hipEventDestroy(e_start));
    CHECK(hipEventDestroy(e_stop));
    CHECK(hipsparseDestroyDnMat(descr_B));
    CHECK(hipsparseDestroyDnMat(descr_X));
    float ms_avg = ms_total / repeats;
    printf("%s,%dx%d,%12.6f\n", is_coo ? "COO" : "CSR", sys.size, sys.size, ms_avg);
}

void trsm_generic(trsm_system & sys)
{
    hipsparseHandle_t handle;
    CHECK(hipsparseCreate(&handle));

    auto set_tri_attrs = [](hipsparseSpMatDescr_t A){
        auto lower   = HIPSPARSE_FILL_MODE_LOWER;
        auto nonunit = HIPSPARSE_DIAG_TYPE_NON_UNIT;
        CHECK(hipsparseSpMatSetAttribute(A, HIPSPARSE_SPMAT_FILL_MODE, &lower,   sizeof(lower)));
        CHECK(hipsparseSpMatSetAttribute(A, HIPSPARSE_SPMAT_DIAG_TYPE, &nonunit, sizeof(nonunit)));
    };
    // CSR
    hipsparseSpMatDescr_t A_Csr;
    CHECK(hipsparseCreateCsr(&A_Csr, sys.size, sys.size, sys.nnz,
                                    sys.A_rowptrs, sys.A_colidxs, sys.A_vals,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    set_tri_attrs(A_Csr);

    // COO
    int *A_coo_rows = nullptr;
    CHECK(hipMalloc(&A_coo_rows, sys.nnz * sizeof(int)));
    CHECK(hipsparseXcsr2coo(handle, sys.A_rowptrs, sys.nnz, sys.size,
                           A_coo_rows, HIPSPARSE_INDEX_BASE_ZERO));

    hipsparseSpMatDescr_t A_coo;
    CHECK(hipsparseCreateCoo(&A_coo, sys.size, sys.size, sys.nnz,
                            A_coo_rows, sys.A_colidxs, sys.A_vals,
                            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    set_tri_attrs(A_coo);

    // Ejecuto los dos formatos
    run_spm(sys, A_Csr, handle);
    run_spm(sys, A_coo, handle, true);

    CHECK(hipsparseDestroySpMat(A_Csr));
    CHECK(hipsparseDestroySpMat(A_coo));
    CHECK(hipFree(A_coo_rows));
    CHECK(hipsparseDestroy(handle));
}

void clear_system(trsm_system & sys)
{
    CHECK(hipFree(sys.A_rowptrs));
    CHECK(hipFree(sys.A_colidxs));
    CHECK(hipFree(sys.A_vals));
    CHECK(hipFree(sys.B));
    CHECK(hipFree(sys.X));
}


int main(){
    std::vector<const char*> matrix_files = {"matrix13.txt", "matrix16.txt", "matrix20.txt", "matrix25.txt"};
    printf("type,size,time\n");
    for(const char* file : matrix_files){
        trsm_system sys = load_system(file);
        trsm_generic(sys);
        clear_system(sys);
    }

    return 0;
}